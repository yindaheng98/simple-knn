#include "hip/hip_runtime.h"
/*
 * Copyright (C) 2023, Inria
 * GRAPHDECO research group, https://team.inria.fr/graphdeco
 * All rights reserved.
 *
 * This software is free for non-commercial, research and evaluation use 
 * under the terms of the LICENSE.md file.
 *
 * For inquiries contact  george.drettakis@inria.fr
 */

#include "spatial.h"
#include "simple_knn.h"

std::tuple<torch::Tensor, torch::Tensor>
KNN(const torch::Tensor& points)
{
  const int P = points.size(0);

  auto int_opts = points.options().dtype(torch::kInt32);
  torch::Tensor idx = torch::full({P, 3}, -1, int_opts);
  auto float_opts = points.options().dtype(torch::kFloat32);
  torch::Tensor dist = torch::full({P, 3}, 0.0, float_opts);
  
  SimpleKNN::knn(P, (float3*)points.contiguous().data<float>(), idx.contiguous().data<int>(), dist.contiguous().data<float>());

  return std::make_tuple(dist, idx);
}